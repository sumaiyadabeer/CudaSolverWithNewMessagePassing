#include "hip/hip_runtime.h"
#include "../inc/calculate_error.h"
#include "../inc/eta_function.h"


__global__ void Lx_b(int *row_ptr,  int *col_off, int *values, float *b, float *b_sink, float *beta, int *L, float *x, float *ans, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;

	//construction of L matrix : use construction of 2d matrix in 1D using macros

	//why not initilizing L to 0s

	//diagonal element D
	int degree = row_ptr[index+1]-row_ptr[index];
	L[(index*n)+index] = degree;

	//admat elelemts -A
	for(int i = row_ptr[index]; i<row_ptr[index+1]; i++){
		L[(col_off[i])+n*(index)]= -values[i];
	//	L[(col_off[i])*n+index]=values[i]; Be careful for undirected graph here
	}

	//calculation of  as ans
	ans[index]=0;
	// if (index == 1)
	// {
		for(int i = n*index;i<n*(index+1);i++){ 
			// printf(" ********** %d \t %d\t %f\n",i,L[i],x[i%n]);
			ans[index] += L[i]*x[(i%n)];//(b[index]/beta[0]*(-b_sink[0]));
			/*if(index==1){`
				printf("%d \t %d\t %f\n",i,L[i],x[i%n]);
			}*/
		}
	// }

	
	// printf("...%d\t%f \t %f\n",index, (b[index]*(-b_sink[0]/beta[0])), b[index]);
	//calculation of Lx-b
	ans[index] = ans[index] - (b[index]); //// because b = \beta*J (b[index]*degree);  (b[index])*((-b_sink[0]/beta[0]));
	//calculate norm of that lx-b //can call eta_norm functions here


}


