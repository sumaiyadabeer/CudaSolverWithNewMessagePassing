#include "hip/hip_runtime.h"
#include "../inc/communication.h"

#include <cassert>


//#define NDEBUG

#ifndef NDEBUG
#define ASSERT_EX(condition, statement) \
    do { \
        if (!(condition)) { statement; assert(condition); } \
    } while (false)
#else
#define ASSERT_EX(condition, statement) ((void)0)
#endif

using namespace std;

__global__ void send(int *row_ptr, float *b, int *col_off, int *values, int *queue, int *outbox, int *cnt, hiprandState *my_curandstate, int seed, int seed2, int E){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	float rand;
	//move queue length to register for each thread
	int Q; //define that in Shared mem
	// if(b[index]>=0.0)
	Q = queue[index];
	outbox[index] = -1; //remove this in simplest recv
	

//printf("%f \t", b[index]);


//generate packet according b

hiprand_init(seed, index, 0, &my_curandstate[index]);
rand = hiprand_uniform(my_curandstate+index);
if(b[index] > 0){
	if(rand <= b[index])
		{
			Q++;
			// printf("[P %d] ", index);				
		}
}	

	

//select random neighbour based on weight 
if (Q>0 && b[index]>=0.0){
	int neighbour;
	int degree = row_ptr[index+1]-row_ptr[index];
	hiprand_init(seed2, index, 0, &my_curandstate[index]);
	rand = hiprand_uniform(my_curandstate);
	rand = rand*degree;
	rand = int(floorf(rand));
	if ( rand >= degree ){
		rand = degree - 1;
	}


	ASSERT_EX( int(row_ptr[index]+(int(rand))) < row_ptr[index+1], 
	printf("index %d: neighbour number = %d with col_off index %d \t degree is  = %d \t current row_ptr is = %d \t next row_ptr is = %d\n",
	index,  int(rand) , int(row_ptr[index]+(floorf(rand))), degree, row_ptr[index],  row_ptr[index+1])
	);
	
	//ASSERT_EX(int(row_ptr[index]+(floor(rand))) < row_ptr[index+1], printf("index %d: neighbour number = %d with col_off index %d \t degree is  = %d \t next row_ptr is = %d\n",index,  int(rand) , int(row_ptr[index]+(floor(rand))), (row_ptr[index+1]-row_ptr[index]), row_ptr[index+1]));
	// assert( (row_ptr[index]+(floor(rand))) < row_ptr[index+1] ); // to chk actual neighbour is selected
	neighbour = col_off[row_ptr[index]+int((floorf(rand)))];
	// printf(" [%d -> %d] ", index, neighbour );
	//write that neighbour to outbox 
	outbox[index] = neighbour;
	//packet sent is subtracted
	Q = Q-1; 
	cnt[index] = cnt[index]+1;
}



//upate the queue value inreg
	queue[index] = Q;
	
	

	//copy reg value to device memory

	
}


//Simplest rcv 
__global__ void recv(int *outbox, int *queue, float *b, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int Q = 0;
	for(int i=0; i<n; i++){ //every thread is scanning whole outbox
		if(outbox[i] == index){
			Q++;
		}
	}

	queue[index] = queue[index] + Q;
	//this might be creating the problem
	// outbox[index] = -1;
	
	//set here the index of sink to whatever now setting as n-1
	// if(b[index]<0){
	// 	queue[index]=0;
	// }
	// //copy the queue to shared mem/reg
	//define shared memory and do looping to get one out box and increment the associated packet to shared mem
	//update the queue value to reg by shared mem 
	//chk eta_t - eta_t-1<epsilon or not

} 

// Thrust Recv
__global__ void thrust_recv(int *outbox_count, int *queue, int *outbox_index, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (outbox_index[index] != -1){
		// printf("%d: copying %d packets to %d\n", index, outbox_count[index], outbox_index[index]);
		queue[outbox_index[index]] = queue[outbox_index[index]] + outbox_count[index];
	}
// outbox[index] = -1;
// outbox_index[index] = -1;

} 

