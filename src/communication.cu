#include "hip/hip_runtime.h"
#include "../inc/communication.h"

#include <cassert>


//#define NDEBUG

#ifndef NDEBUG
#define ASSERT_EX(condition, statement) \
    do { \
        if (!(condition)) { statement; assert(condition); } \
    } while (false)
#else
#define ASSERT_EX(condition, statement) ((void)0)
#endif

using namespace std;

__global__ void send(int *row_ptr, float *b, int *col_off, int *values, int *queue, int *outbox, int *cnt, hiprandState *my_curandstate, int seed, int seed2, int E){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	float rand = hiprand_uniform(my_curandstate+index);
	//move queue length to register for each thread
	int Q = 0; //define that in Shared mem
	// if(b[index]>=0.0)
	Q = queue[index];
	

//printf("%f \t", b[index]);


//generate packet according b

hiprand_init(seed, index, 0, &my_curandstate[index]);
rand = hiprand_uniform(my_curandstate);
if(b[index] > 0){
	if(rand <= b[index])
		{
			Q++;
			// printf("[P %d] ", index);				
		}
}	

	

//select random neighbour based on weight 
if (Q>0 && b[index]>=0.0){
	int neighbour;
	hiprand_init(seed2, index, 0, &my_curandstate[index]);
	rand = hiprand_uniform(my_curandstate);
	rand = rand*(row_ptr[index+1]-row_ptr[index]);

	
	ASSERT_EX((row_ptr[index]+(floor(rand))) < row_ptr[index+1], printf("neighbour number selected = %d \t degree is  = %d \n", int(rand) ,  (row_ptr[index+1]-row_ptr[index])));
	assert( (row_ptr[index]+(floor(rand))) < row_ptr[index+1] ); // to chk actual neighbour is selected
	neighbour = col_off[row_ptr[index]+int(floor(rand))];
	// printf(" [%d -> %d] ", index, neighbour );
	//write that neighbour to outbox 
	outbox[index] = neighbour;
	//packet sent is subtracted
	Q = Q-1; 
	cnt[index] = cnt[index]+1;
}



//upate the queue value inreg
	queue[index] = Q;
	
	

	//copy reg value to device memory
	
}


//Simplest rcv 
__global__ void recv(int *outbox, int *queue, float *b, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int Q = 0;
	for(int i=0; i<n; i++){ //every thread is scanning whole outbox
		if(outbox[i] == index){
			Q++;
		}
	}

	queue[index] = queue[index] + Q;
	outbox[index] = -1;
	//set here the index of sink to whatever now setting as n-1
	// if(b[index]<0){
	// 	queue[index]=0;
	// }
	// //copy the queue to shared mem/reg
	//define shared memory and do looping to get one out box and increment the associated packet to shared mem
	//update the queue value to reg by shared mem 
	//chk eta_t - eta_t-1<epsilon or not

} 

