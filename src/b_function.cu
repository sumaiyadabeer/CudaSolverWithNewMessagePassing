#include "hip/hip_runtime.h"
#include "../inc/b_function.h"


__global__ void initialize(float *eta, int *cnt, int *queue, int *outbox, int *L, int n, int e, hiprandState *state, int rand){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
	// for (int i=0;i<n;i++){
	// 	L[n*(index)+i]=0;
	// }    
        
        hiprand_init(rand, index, 0, &state[index]);
        float randd = hiprand_uniform(state+index);
        // queue[index]= 0;
        queue[index]= int((3)*randd);
        outbox[index]=-1;
        cnt[index]=0;
        eta[index]=0;
        //define N and E in const memory
        //define beta globaly to 1

}

__global__ void convert_b_to_J(float *b,   int n, float *sink_b){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        //normalization of b 
        //b[index]=b[index]/(-b[n-1]);
        b[index] = b[index]/(-sink_b[0]); 
}
__global__ void convert_J_to_2betaJ(float *J,   int n, float *beta){
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        //normalization of b 
        //b[index]=b[index]/(-b[n-1]);
        J[index] = 2*J[index]*beta[0]; //multiplied by 2 as we are reducing by 2 in each iter
}

__global__ void update_b( float *b){
        //put beta in const memory
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        b[index] = b[index]/2;
}

// __global__ void calculate_DJ(int *row_ptr, float *J, float *normalized_b ){
// 	int index = threadIdx.x + blockIdx.x * blockDim.x; 
// 	J[index] = normalized_b[index] *(row_ptr[index+1]-row_ptr[index]);
// 	// printf("%d \t %f \n", index, normalized_b[index]);
// }

// this could be done by finding min using atomic max
__global__ void get_b_sink( float *b, float *b_sink, int n, int *sink_index){

        int index = threadIdx.x + blockIdx.x * blockDim.x;
        // b_sink[0]=0.0;
        // __syncthreads();
        if (b[index] < 0.0){
                b_sink[0] = b[index];
                sink_index[0] = index;
                // printf("%d", index);
        }
}