#include "hip/hip_runtime.h"
#include "../inc/eta_function.h"

__global__ void make_cnt_0(int *cnt){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	cnt[index] = 0;
}
__global__ void copy_eta(float *eta, float *eta_mins){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	eta_mins[index]=eta[index];
} 

__global__ void calculate_eta(float *eta, int *cnt, float T){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	eta[index]=(cnt[index])*(1/T);
	//printf("%d \t %d \t %fd \t %f\n", index, cnt[index], T,eta[index]);
} 

__global__ void calculate_etaQ(float *eta, int *Q, float t){
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	float eta_prev = eta[index];

	eta[index] =  (((t-1)*eta_prev) + Q[index])/t; 

	// eta[index]=(cnt[index])*(1/T);
	//printf("%d \t %d \t %fd \t %f\n", index, cnt[index], T,eta[index]);
} 

__global__ void get_eta_max(float *eta,float *eta_max, int n){
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (index==0){
		*eta_max=0.0;
		for(int i=0; i<n;i++){
			if(eta[i]>*eta_max)
				*eta_max=eta[i];
		}
	}
	__syncthreads();
}

__global__ void calculate_eta_del(float *eta_del, float *eta, float *eta_tminus){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	eta_del[index]= eta[index]-eta_tminus[index]; 
}


__global__ void one_norm(float *eta_del, float *one_norm, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	*one_norm=0.0;
	// printf("*****-------------------*******");
	if (index==0){
		for(int i=0; i<n;i++){
			if(eta_del[i]<0){
				*one_norm= *one_norm-eta_del[i];
			}else{
				*one_norm= *one_norm+eta_del[i];
			}	
		}
	 }
}
__global__ void one_norm(int *eta_del, int *one_norm, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	*one_norm=0.0;
	if (index==0){
		for(int i=0; i<n;i++){
			if(eta_del[i]<0){
				*one_norm= *one_norm-eta_del[i];
			}else{
				*one_norm= *one_norm+eta_del[i];
			}	
		}
	 }
}

__global__ void two_norm(float *eta_del,float *two_norm, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// *two_norm=0.0;
	// printf("%d \t %f \n", n, eta_del[index]);

	eta_del[index] = eta_del[index]*eta_del[index];
	__syncthreads();
	// if (index==0){
	// 	*two_norm = 0.1;
	// 	for(int i=0; i<n;i++){
	// 		*two_norm = *two_norm + eta_del[i];
	// 		// printf("%f", *two_norm); // DONT KNOW WHY REMOVING THIS LINE MAKES EVERYTHING ZERO !!!!!!!!
	// 	}
	// printf(" \n");
	// *two_norm = sqrt(*two_norm);
	// // printf("%f -- \n", two_norm[0]);
	// }

}

__global__ void infinity_norm(float *eta_del,float *infi_norm, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	*infi_norm=0.0;
	if (index==0){
		for(int i=0; i<n;i++){
			if(abs(eta_del[i])>(*infi_norm)){
				*infi_norm= eta_del[i];
			}		
		}
	 }
}


