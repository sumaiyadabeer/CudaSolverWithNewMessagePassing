#include "hip/hip_runtime.h"
#include<stdio.h>
#include <string>
#include <stdlib.h>
#include<time.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>

#include <algorithm>
#include <iterator>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>
#include <chrono>

#include "../inc/b_function.h"
#include "../inc/communication.h"
#include "../inc/eta_function.h"
#include "../inc/helper.h"
#include "../inc/calculate_error.h"

// #define N 16
// #define E 48
#define THREADS_PER_BLOCK 32

#include <hip/hip_runtime.h>
#include "hipblas.h"
#include ""

using namespace std::chrono;

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    printf("%s\n",hipGetErrorString(x)); \
    system("pause"); \
    return EXIT_FAILURE;}} while(0)


__global__ void solve(int *row_ptr, float *b_sink, float *eta, float *beta, float kappa, float *x ){
	int index = threadIdx.x + blockIdx.x * blockDim.x; 
	//printf("%f \n", eta[index]/(row_ptr[index+1]-row_ptr[index]));	
	//chk kappa
	// x[index]=(-b_sink[0]/beta[0])*(eta[index]/(row_ptr[index+1]-row_ptr[index]));
	x[index]=(eta[index]/(row_ptr[index+1]-row_ptr[index])); 
}



__global__ void solve_scale(int *row_ptr, float *b, float *b_sink, float *eta, float *beta, float kappa, float *x ){
	int index = threadIdx.x + blockIdx.x * blockDim.x; 
	float multiplier=0.0;

	if(abs(b[index]*(-b_sink[0]/beta[0]))>0.0){
		multiplier=(b[index]*(-b_sink[0]/beta[0]));
		printf("%d\t %f \t %f\n",index, b[index], multiplier);
	}
	x[index]=x[index]*multiplier;
	//printf("%f \n", eta[index]/(row_ptr[index+1]-row_ptr[index]));	
	//chk kappa
	//x[index]= (-b_sink/beta)*(eta[index]/(row_ptr[index+1]-row_ptr[index]));
	//x[index]=(-b_sink[0]/beta[0])*(eta[index]/(row_ptr[index+1]-row_ptr[index]));
}

__global__ void solve_shift(int *row_ptr, float *b_sink, float *eta, float *beta, float kappa, float *x ){
	// int index = threadIdx.x + blockIdx.x * blockDim.x; 
	//printf("%f \n", eta[index]/(row_ptr[index+1]-row_ptr[index]));	
	//chk kappa
	//x[index]= (-b_sink/beta)*(eta[index]/(row_ptr[index+1]-row_ptr[index]));
	//x[index]=(-b_sink[0]/beta[0])*(eta[index]/(row_ptr[index+1]-row_ptr[index]));
}

int cublas_two_norm(int N, float *vector, float *norm){
	hipblasStatus_t stat;
    hipblasHandle_t handle;
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("%d \n", stat);
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    // calculate_DJ<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr, d_J, d_b); 
    stat = hipblasSnrm2(handle, N , vector, 1, norm);
    if(stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("norm is not calculated using Cublas\n");
        hipFree (vector);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
   
	return EXIT_SUCCESS;
}



int main(void) {
	
	int devNum = -1;
    CUDA_CALL(hipGetDevice(&devNum));
    CUDA_CALL(hipSetDevice(devNum));
	printf("Code is executing on device %d \n",devNum );
	// return 0;


	// device copies 
	int *d_row_ptr, *d_col_off, *d_values, *d_b_sum, *d_L; 
	int *d_queue, *d_outbox, *d_cnt, *d_stable_cnt, *d_b_sink_index, *d_sum_Q ;
	float  *d_eta, *d_eta_tminusone, *d_eta_del, *d_eta_max, *d_eta_sum, *d_eta_del_norm;
	float *d_b, *d_J, *d_b_norm, *d_b_sink,  *d_x, *d_Lx_b, *d_Lx_b_norm, *d_beta;
	hiprandState *d_state;



	std::string file_path="./generated_input.txt";
	std::string answer_file_path="./generated_answer.txt";
	int NE[2];
	read_file_by_line(file_path, NE, 0, 2);
	const unsigned int N = NE[0];
	const unsigned int E = NE[1]; //BECAUSE IN UNDIRECTED GRAPH EVERY EDGE IS COUNTED TWICE
	

// Alloc space for device copies of graph N E and beta epsilon and kappa eta max
	const unsigned int int_size =  sizeof(int);
	const unsigned int float_size = sizeof(float);
	

	CUDA_CALL(hipMalloc((void **)&d_row_ptr, (N+1)*int_size));
	CUDA_CALL(hipMalloc((void **)&d_b, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_J, N*float_size));  
	CUDA_CALL(hipMalloc((void **)&d_b_norm, float_size));
	CUDA_CALL(hipMalloc((void **)&d_x, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_Lx_b, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_Lx_b_norm, float_size));

	CUDA_CALL(hipMalloc((void **)&d_eta_sum, float_size));
	CUDA_CALL(hipMalloc((void **)&d_eta_max, float_size));
	CUDA_CALL(hipMalloc((void **)&d_eta_del_norm, float_size));
	CUDA_CALL(hipMalloc((void **)&d_eta, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_eta_del, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_eta_tminusone, N*float_size));
	CUDA_CALL(hipMalloc((void **)&d_b_sink, float_size));
	CUDA_CALL(hipMalloc((void **)&d_b_sink_index, int_size));
	CUDA_CALL(hipMalloc((void **)&d_sum_Q, int_size));
	CUDA_CALL(hipMalloc((void **)&d_beta, float_size));

	CUDA_CALL(hipMalloc((void **)&d_col_off, E*int_size));
	CUDA_CALL(hipMalloc((void **)&d_values, E*int_size));
	CUDA_CALL(hipMalloc((void **)&d_b_sum, int_size));

	CUDA_CALL(hipMalloc((void **)&d_queue, N*int_size));
	CUDA_CALL(hipMalloc((void **)&d_outbox, N*int_size));
	CUDA_CALL(hipMalloc((void **)&d_cnt, N*int_size));
	CUDA_CALL(hipMalloc((void **)&d_stable_cnt, N*int_size));
	CUDA_CALL(hipMalloc((void **)&d_L, N*N*int_size));

	CUDA_CALL(hipMalloc(&d_state, N*sizeof(hiprandState)));
	

	//read the graph and b from input file

	int *row_ptr = (int*)malloc((N+1)*int_size);
	read_file_by_line(file_path, row_ptr, 1, N+1);
	
	int *col_off = (int*)malloc(E*int_size);
	read_file_by_line(file_path, col_off, 2, E);

	int *values = (int*)malloc(E*int_size);
	read_file_by_line(file_path, values, 3, E);

	float *b = (float*)malloc(N*float_size);
	read_file_by_line(file_path, b, 4, N);

	// float *jacobi = (float*)malloc((N)*float_size);
	// read_file_by_line(answer_file_path, jacobi, 1, N);

	//assert in function "read_file_by_line" if input is less than N+1/N/E not working 

	

	// // this loop is for printing purpose of input values
	// for (int i=0;i<N;i++){
	// 	std :: cout<<i<<"\t"<< jacobi[i]<<std :: endl;
	// }
	// return -1;
	

// Copy inputs to device
	CUDA_CALL(hipMemcpy(d_row_ptr, row_ptr, (N+1)*int_size, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(d_col_off, col_off, E*int_size, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(d_values, values, E*int_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(d_b, b, N*float_size, hipMemcpyHostToDevice));
	// CUDA_CALL(hipMemcpy(d_DJ, b, N*float_size, hipMemcpyHostToDevice));
 
	printf("input and copy to device done \n");
// Host space allocation
	float *eta = (float*)malloc(N*float_size);
	int *queue = (int*)malloc(N*int_size);
	float *rhs_norm = (float*)malloc(float_size);
	float *Lx_b_norm = (float*)malloc(N*float_size);
	float *beta = (float*)malloc(float_size);
	float *result = (float*)malloc(N*sizeof(float));
	float *eta_del_norm = (float*)malloc(float_size); 
	float *eta_sum = (float*)malloc(float_size);
	float *eta_max = (float*)malloc(float_size);
	
	
//Initial_setup
	
	const double EPS =  1.19209e-07; //1.0/(N*N*N);
	double eta_max_threshold = 0.9; //(0.75)*(1-EPS);	see the logic in paper
	float frac_of_packet_sunk_threshold = 0.9;

	int num_of_blocks =  (N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;  
	unsigned int max_epoch	= 100000; //100000; //should depend on graph size and topology
	unsigned int epoch, stable_epoch;
	int sink_index;
	int Q_sink_index;
	int sum_Q;
	float frac_of_packet_sunk;
	int eta_gt_chk_more_thn_i;
	int eta_del_lt_eps_more_thn_i;
	int frac_of_packet_sunk_more_thn_i;
	float send_recv_rounds;
	bool flag_frac_of_packet = false;

	

	*beta = 1.0;
	CUDA_CALL(hipMemcpy(d_beta, beta, float_size, hipMemcpyHostToDevice));

	
	get_b_sink<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_b, d_b_sink, N, d_b_sink_index); // return min vale of sink (-9)
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipMemcpy(&sink_index, d_b_sink_index, int_size, hipMemcpyDeviceToHost));
	

	convert_b_to_J<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_b, N, d_b_sink);
	CUDA_CALL(hipDeviceSynchronize());
	CUDA_CALL(hipMemcpy(b, d_b, N*float_size, hipMemcpyDeviceToHost));
	convert_J_to_2betaJ<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_b, N, d_beta); //bcz this will get halved after entering the loop
	CUDA_CALL(hipDeviceSynchronize());

	//this is calculation of 2 norm of ||DJ||_2 using cublas

    // hipblasStatus_t stat;
    // hipblasHandle_t handle;
    // stat = hipblasCreate(&handle);
    // if (stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("CUBLAS initialization failed\n");
    //     return EXIT_FAILURE;
    // }
    // // calculate_DJ<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr, d_J, d_b); 
    // stat = hipblasSnrm2(handle, N , d_b, 1, rhs_norm);
    // if(stat != HIPBLAS_STATUS_SUCCESS) {
    //     printf ("b norm is not calculated using Cublas");
    //     hipFree (d_b);
    //     hipblasDestroy(handle);
    //     return EXIT_FAILURE;
    // }
	// printf("%f \n", *rhs_norm);
	// *rhs_norm = 0.0;

	// printf("%d", cublas_two_norm( N, d_b, rhs_norm));
	// printf("%f \n", *rhs_norm);

	
	do{
		high_resolution_clock::time_point t1 = high_resolution_clock::now();
  		// *beta = *beta/2;
		send_recv_rounds = *beta;
		eta_gt_chk_more_thn_i = 0;
		frac_of_packet_sunk_more_thn_i = 0;
		eta_del_lt_eps_more_thn_i = 0;
		epoch = 0;
		stable_epoch = 0;

		while (send_recv_rounds < 10.0) //this loop is to make sure to generate packet in group of epoch.. waz ctraeting problem inn visualization
			send_recv_rounds *= 10.0;
				
		CUDA_CALL(hipMemcpy(d_beta, beta, sizeof(float), hipMemcpyHostToDevice));
		update_b<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_b);
		// CUDA_CALL(hipMemcpy(b, d_b, N*sizeof(float), hipMemcpyDeviceToHost));
		// CUDA_CALL(hipDeviceSynchronize());
		// for (int i=0; i<N; i++)
		// 	printf("%f\t", b[i]);
		// printf("\n");

		initialize<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_cnt, d_queue, d_outbox, d_L, N, E, d_state, 2*rand()); //set eta queue outbox as 0_randomstae
		// initialize<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_stable_cnt, d_queue, d_outbox, d_L, N, E, d_state, 2*rand()); //set eta queue outbox as 0_randomstae
		
		do{
			get_b_sink<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_b, d_b_sink, N, d_b_sink_index);
			CUDA_CALL(hipMemcpy(beta, d_b_sink, sizeof(float), hipMemcpyDeviceToHost));
			*beta = -*beta;
			printf("In DRW compute iter: %d beta: %f \n", epoch, *beta);
			copy_eta<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_eta_tminusone);
			CUDA_CALL(hipDeviceSynchronize());

			for(int i=0; i<(int)send_recv_rounds; i++){
				epoch++;
				stable_epoch++;
				// printf("%d ", epoch);
				send<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr, d_b, d_col_off, d_values, d_queue, d_outbox, d_cnt, d_state,  rand(), rand(), E);
				CUDA_CALL(hipDeviceSynchronize());
				recv<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_outbox, d_queue, d_b, N);
				CUDA_CALL(hipDeviceSynchronize());
				
				// CUDA_CALL(hipMemcpy(queue, d_queue,(N)*sizeof(int), hipMemcpyDeviceToHost));
				// printf("\nprinting queues: \t");
				// for (int i=0; i<N; i++)
				// 	printf("%d\t", queue[i]);
				// printf("\n");
			}
			// if (flag_frac_of_packet == false){
				calculate_eta<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_cnt, float(epoch));
				CUDA_CALL(hipDeviceSynchronize());
				CUDA_CALL(hipMemcpy(eta,d_eta,(N)*sizeof(float), hipMemcpyDeviceToHost));
			// }else{
			// 	calculate_eta<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_cnt, float(stable_epoch));
			// 	CUDA_CALL(hipDeviceSynchronize());
			// 	CUDA_CALL(hipMemcpy(eta,d_eta,(N)*sizeof(float), hipMemcpyDeviceToHost));
			// }


			printf("\nprinting eta \n");
			for (int i=0; i<N; i++)
				printf("%f\n", eta[i]);
			printf("printing eta ends \n");

			solve<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr, d_b_sink, d_eta,  d_beta, 0.0001 , d_x );//(int *row_ptr, float *b_sink, float *eta, float *beta, float kappa, float *x )
			CUDA_CALL(hipDeviceSynchronize());			
			CUDA_CALL(hipMemcpy(eta, d_x, (N)*sizeof(
				float), hipMemcpyDeviceToHost));

			printf("\nprinting x \n");
			for (int i=0; i<N; i++)
				printf("%f\n", eta[i]);
			printf("printing x ends \n");
			//call kernel for x-x' // later when calculation of x got some speed
			//print x-x' for each coordinate // later when calculation of x got some speed

			Lx_b<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr,  d_col_off, d_values, d_b, d_b_sink, d_beta, d_L, d_x, d_Lx_b, N);
			CUDA_CALL(hipDeviceSynchronize());
			CUDA_CALL(hipMemcpy(result, d_Lx_b, N*sizeof(float), hipMemcpyDeviceToHost));


			printf("printing Lx-b \n");
			for (int i=0;i<N;i++)
				printf("%d \t %f \n",i, result[i]);
			printf("printing Lx-b ends\n");

			
			two_norm<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_Lx_b,d_Lx_b_norm, N);
			one_norm<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_Lx_b,d_Lx_b_norm, N);
			CUDA_CALL(hipDeviceSynchronize());
			CUDA_CALL(hipMemcpy(Lx_b_norm, d_Lx_b_norm, sizeof(float), hipMemcpyDeviceToHost));
			// if (cublas_two_norm( N, d_Lx_b, Lx_b_norm) == EXIT_SUCCESS){
			// 	printf("two norm of Lx-b is calculated using cublas \n");				
			// }else{
			// 	printf("cublas is not calculating norm for Lx-b \n");
			// 	return -1;
			// }
			*Lx_b_norm = sqrt(*Lx_b_norm);
			std::cout<<"Error is " << (*Lx_b_norm)<<std::endl; //<< should be (*Lx_b_norm)/((*beta)*(*rhs_norm)) for actual error
			//call two norm of x-x' here // later when calculation of x got some speed
			//print two norm of x-x' here // later when calculation of x got some speed
			

			// int *LapMat = (int*)malloc(N*N*sizeof(int));
			// hipMemcpy(LapMat, d_L, N*N*sizeof(int), hipMemcpyDeviceToHost);                
			// printf("printing LapMat \n");
			// 		for (int i=0; i<N*N; i++){
			// 				printf("%d\t", LapMat[i]);
			// if((i+1)%N==0)
			// 	printf("\n");
			// }
			// printf("printing LapMat ends\n");

			/**************Termination condition prep based on eta del *******************/
			calculate_eta_del<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta_del,d_eta, d_eta_tminusone);
			CUDA_CALL(hipDeviceSynchronize());
			// if (cublas_two_norm( N, d_eta_del, eta_del_norm) == EXIT_SUCCESS){	
			// 	printf("two norm of eta is calculated using cublas \n");						
			// }else{
			// 	printf("cublas is not calculating norm for eta\n");
			// 	return -1;
			// }
			two_norm<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta_del,d_eta_del_norm, N);
			one_norm<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta_del,d_eta_del_norm, N);
			CUDA_CALL(hipDeviceSynchronize());
			CUDA_CALL(hipMemcpy(eta_del_norm, d_eta_del_norm, sizeof(float), hipMemcpyDeviceToHost));
			*eta_del_norm = sqrt(*eta_del_norm);
			// solve<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_row_ptr, d_b_sink, d_eta,  d_beta, 0.0001 , d_x );
			// hipDeviceSynchronize();

			/**************Termination condition prep if queues are saturated *******************/
			get_eta_max<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_eta_max, N);
			CUDA_CALL(hipDeviceSynchronize());
			CUDA_CALL(hipMemcpy(eta_max, d_eta_max, sizeof(float), hipMemcpyDeviceToHost));	
			printf("Epoch: %d \t eta_del_norm: %f \t eta_del_norm<=EPS: %s \t eta_del_norm>0: %s \t eta_max_inner: %f\n", epoch, *eta_del_norm, (*(eta_del_norm) <= EPS)?"T":"F", (*(eta_del_norm)>0)?"T":"F", *eta_max);
			/**************Termination condition prep for Q[sink]/(1+sum(Q)) *******************/
	

			CUDA_CALL(hipMemcpy(&Q_sink_index, d_queue+sink_index, int_size, hipMemcpyDeviceToHost));
			printf("sink index : %d \n", sink_index );
			one_norm<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_queue, d_sum_Q, N);
			CUDA_CALL(hipMemcpy(&sum_Q, d_sum_Q, int_size, hipMemcpyDeviceToHost)); //has some issues
			frac_of_packet_sunk = (float)Q_sink_index/(float)(1+sum_Q);
			printf("**************%f\t %d\t %d\n", frac_of_packet_sunk, Q_sink_index, sum_Q);

			

			// Termination in action	
			if(((*(eta_del_norm) <= EPS) && (*(eta_del_norm)>0))){
				get_eta_max<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_eta_max, N); // these three lines are going to be used after exiting the loop 
				CUDA_CALL(hipDeviceSynchronize());
				CUDA_CALL(hipMemcpy(eta_max, d_eta_max, sizeof(float), hipMemcpyDeviceToHost));

				eta_del_lt_eps_more_thn_i++;
				if (eta_del_lt_eps_more_thn_i >= 10){
					printf("eta_del_norm is lt threshold so breaking\n");
					break;
				}
			}else if((*eta_max > eta_max_threshold)){
				get_eta_max<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_eta_max, N); // these three lines are going to be used after exiting the loop 
				CUDA_CALL(hipDeviceSynchronize());
				CUDA_CALL(hipMemcpy(eta_max, d_eta_max, sizeof(float), hipMemcpyDeviceToHost));
				
				eta_gt_chk_more_thn_i++;
				if (eta_gt_chk_more_thn_i >= 10){
					printf("eta_max is gt threshold so breaking\n");
					break;
				}

			}else if(frac_of_packet_sunk > frac_of_packet_sunk_threshold) {
				get_eta_max<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_eta, d_eta_max, N); // these three lines are going to be used after exiting the loop 
				CUDA_CALL(hipDeviceSynchronize());
				CUDA_CALL(hipMemcpy(eta_max, d_eta_max, sizeof(float), hipMemcpyDeviceToHost));
				
				frac_of_packet_sunk_more_thn_i++;
				if (flag_frac_of_packet != true){
					if (frac_of_packet_sunk_more_thn_i >= 10){ 
						flag_frac_of_packet = true;
						stable_epoch = 0;
						make_cnt_0<<<num_of_blocks,THREADS_PER_BLOCK>>>(d_cnt);
						CUDA_CALL(hipDeviceSynchronize());
						printf("frac_of_packet_sunk is gt threshold so breaking\n");						
						break; 
					}
				}
					

			}else{
				eta_gt_chk_more_thn_i = 0;
				frac_of_packet_sunk_more_thn_i = 0;
			}

		}while(epoch < max_epoch);
		high_resolution_clock::time_point t2 = high_resolution_clock::now();
  		duration<double> time_span = duration_cast<duration<double>>(t2 - t1);

		printf("epoch: %d \t eta_max: %f \t max_allowed_eta: %f \t duartion: ", epoch, *eta_max, eta_max_threshold);
		
		std::cout<<time_span.count()<<" sec"<<std::endl;
		
		// printf("x_solve.py*beta: ");
		// for(int i=0; i<N; i++)
		// 	printf("%f \t", (jacobi[i])*(*beta));
		// printf("\n");

	}while((*eta_max) > eta_max_threshold && (*eta_max)>0);
	

// Cleanup
	free(row_ptr);    free(b);    free(eta);    free(col_off);    free(values);    free(rhs_norm);	free(Lx_b_norm);	free(beta);    free(result); 	free(eta_del_norm); 	free(eta_sum); 	  free(eta_max);
	hipFree(d_row_ptr); 	hipFree(d_b); 	hipFree(d_J); 	hipFree(d_b_norm); 	hipFree(d_x); 	hipFree(d_Lx_b); 	hipFree(d_Lx_b_norm);	hipFree(d_eta_sum);	hipFree(d_eta_max); 	hipFree(d_eta_del_norm); 	hipFree(d_eta); 	hipFree(d_eta_del);	hipFree(d_eta_tminusone); 	hipFree(d_b_sink); 	hipFree(d_b_sink_index);	hipFree(d_beta);	hipFree(d_col_off); 	hipFree(d_values); 	hipFree(d_b_sum);	hipFree(d_queue); 	hipFree(d_outbox);	hipFree(d_cnt);	hipFree(d_L);	hipFree(&d_state);
	return 0;
}

